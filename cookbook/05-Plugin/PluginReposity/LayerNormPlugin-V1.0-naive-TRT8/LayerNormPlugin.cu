#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "LayerNormPlugin.h"

__global__ void layerNormKernel(float *pInput, float *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * 256 + threadIdx.x;

    __shared__ float temp[128];

    float value0 = pInput[index];
    float value1 = pInput[index + 128];
    temp[tx]     = value0 + value1;
    __syncthreads();

    for (int stride = 64; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float mean = temp[0] / 256;
    __syncthreads();

    temp[tx] = (value0 - mean) * (value0 - mean) + (value1 - mean) * (value1 - mean);
    __syncthreads();

    for (int stride = 64; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float var = temp[0] / 256;

    pOutput[index]       = (value0 - mean) * rsqrtf(var + EPSILON);
    pOutput[index + 128] = (value1 - mean) * rsqrtf(var + EPSILON);
}

namespace nvinfer1
{
// class LayerNormPlugin
LayerNormPlugin::LayerNormPlugin(const std::string &name):
    name_(name)
{
    WHERE_AM_I();
}

LayerNormPlugin::LayerNormPlugin(const std::string &name, const void *buffer, size_t length):
    name_(name)
{
    WHERE_AM_I();
}

LayerNormPlugin::~LayerNormPlugin()
{
    WHERE_AM_I();
}

IPluginV2DynamicExt *LayerNormPlugin::clone() const noexcept
{
    WHERE_AM_I();
    auto p = new LayerNormPlugin(name_, nullptr, 0);
    p->setPluginNamespace(namespace_.c_str());
    return p;
}

int32_t LayerNormPlugin::getNbOutputs() const noexcept
{
    WHERE_AM_I();
    return 1;
}

DataType LayerNormPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
{
    WHERE_AM_I();
    return DataType::kFLOAT;
}

DimsExprs LayerNormPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
{
    WHERE_AM_I();
    return inputs[0];
}

bool LayerNormPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    switch (pos)
    {
    case 0:
        return inOut[0].type == DataType::kFLOAT && inOut[0].format == TensorFormat::kLINEAR;
    case 1:
        return inOut[1].type == inOut[0].type && inOut[1].format == inOut[0].format;
    default: // should NOT be here!
        return false;
    }
    return false;
}

void LayerNormPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept
{
    WHERE_AM_I();
    return 0;
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    const int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1]; // 仅用于处理 nEmbedding 为 256 的情况

    layerNormKernel<<<nBlock, 128, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
    return 0;
}

void LayerNormPlugin::destroy() noexcept
{
    WHERE_AM_I();
    delete this;
    return;
}

int32_t LayerNormPlugin::initialize() noexcept
{
    WHERE_AM_I();
    return 0;
}

void LayerNormPlugin::terminate() noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPlugin::getSerializationSize() const noexcept
{
    WHERE_AM_I();
    return 0;
}

void LayerNormPlugin::serialize(void *buffer) const noexcept
{
    WHERE_AM_I();
    return;
}

void LayerNormPlugin::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPlugin::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPlugin::getPluginType() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPlugin::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

void LayerNormPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
{
    WHERE_AM_I();
    return;
}

void LayerNormPlugin::detachFromContext() noexcept
{
    WHERE_AM_I();
    return;
}

// class LayerNormPluginCreator
PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

LayerNormPluginCreator::LayerNormPluginCreator()
{
    WHERE_AM_I();
    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

LayerNormPluginCreator::~LayerNormPluginCreator()
{
    WHERE_AM_I();
}

IPluginV2 *LayerNormPluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    WHERE_AM_I();
    float                          epsilon = 1.0e-5f;
    std::map<std::string, float *> parameterMap {{"epsilon", &epsilon}};

    for (int i = 0; i < fc->nbFields; ++i)
    {
        if (parameterMap.find(fc->fields[i].name) != parameterMap.end())
        {
            *parameterMap[fc->fields[i].name] = *reinterpret_cast<const float *>(fc->fields[i].data);
        }
    }
    return new LayerNormPlugin(name);
}

IPluginV2 *LayerNormPluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    WHERE_AM_I();
    return new LayerNormPlugin(name, serialData, serialLength);
}

void LayerNormPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPluginCreator::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPluginCreator::getPluginName() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPluginCreator::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

const PluginFieldCollection *LayerNormPluginCreator::getFieldNames() noexcept
{
    WHERE_AM_I();
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);

} // namespace nvinfer1
