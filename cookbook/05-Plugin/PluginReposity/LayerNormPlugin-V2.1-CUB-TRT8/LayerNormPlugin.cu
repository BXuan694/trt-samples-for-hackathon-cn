#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "LayerNormPlugin.h"

template<int VPT>
struct BytesToType;

template<>
struct BytesToType<2>
{
    using type = uint16_t;
};
template<>
struct BytesToType<4>
{
    using type = uint32_t;
};
template<>
struct BytesToType<8>
{
    using type = uint64_t;
};
template<>
struct BytesToType<16>
{
    using type = float4;
};

template<int Bytes>
__device__ inline void copy(const void *local, void *data)
{
    using T = typename BytesToType<Bytes>::type;

    const T *in  = static_cast<const T *>(local);
    T *      out = static_cast<T *>(data);
    *out         = *in;
}

struct mySum
{
    __host__ __device__ __forceinline__ float2 operator()(const float2 &a, const float2 &b) const
    {
        return make_float2(a.x + b.x, a.y + b.y);
    }
};

template<typename T, int TPB, int VPT>
__global__ void layerNormKernel(const T *input, const T *gamma, const T *beta, T *output)
{
    const int   idx = blockIdx.x * 256 + VPT * threadIdx.x;
    T           localX[VPT], localGamma[VPT], localBeta[VPT];
    float2      localFloat2 = {0.f, 0.f};
    const float denominator = float(1) / float(256);

    copy<sizeof(T) * VPT>(&input[idx], localX);
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        const float tmp = denominator * (float)localX[it];
        localFloat2.x += tmp;
        localFloat2.y += tmp * (float)localX[it];
    }

    copy<sizeof(T) * VPT>(&gamma[threadIdx.x * VPT], localGamma);
    copy<sizeof(T) * VPT>(&beta[threadIdx.x * VPT], localBeta);

    using BlockReduce = hipcub::BlockReduce<float2, TPB>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float                             mu;     // mean
    __shared__ float                             rsigma; // 1 / std.dev.

    const float2 sumKV = BlockReduce(temp).Reduce(localFloat2, mySum());

    if (threadIdx.x == 0)
    {
        mu     = sumKV.x;
        rsigma = rsqrt(sumKV.y - mu * mu + 1e-6);
    }
    __syncthreads();
#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        localX[it] = ((float)localX[it] - mu) * rsigma * (float)localGamma[it] + (float)localBeta[it];
    }

    copy<sizeof(T) * VPT>(localX, &output[idx]);
}

template __global__ void layerNormKernel<float, 64, 4>(const float *, const float *, const float *, float *);
template __global__ void layerNormKernel<half, 32, 8>(const half *, const half *, const half *, half *);

namespace nvinfer1
{
// class LayerNormPlugin
LayerNormPlugin::LayerNormPlugin(const std::string &name, float epsilon):
    name_(name)
{
    WHERE_AM_I();
    m_.epsilon = epsilon;
}

LayerNormPlugin::LayerNormPlugin(const std::string &name, const void *buffer, size_t length):
    name_(name)
{
    WHERE_AM_I();
    memcpy(&m_, buffer, sizeof(m_));
}

LayerNormPlugin::~LayerNormPlugin()
{
    WHERE_AM_I();
}

IPluginV2DynamicExt *LayerNormPlugin::clone() const noexcept
{
    WHERE_AM_I();
    auto p = new LayerNormPlugin(name_, &m_, sizeof(m_));
    p->setPluginNamespace(namespace_.c_str());
    return p;
}

int32_t LayerNormPlugin::getNbOutputs() const noexcept
{
    WHERE_AM_I();
    return 1;
}

DataType LayerNormPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
{
    WHERE_AM_I();
    return inputTypes[0];
}

DimsExprs LayerNormPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
{
    WHERE_AM_I();
    return inputs[0];
}

bool LayerNormPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    switch (pos)
    {
    case 0:
        return (inOut[0].type == DataType::kFLOAT || inOut[0].type == DataType::kHALF) && inOut[0].format == TensorFormat::kLINEAR;
    case 1:
    case 2:
    case 3:
        return inOut[pos].type == inOut[0].type && inOut[pos].format == inOut[0].format;
    default: // should NOT be here!
        return false;
    }
    return false;
}

void LayerNormPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept
{
    WHERE_AM_I();
    return 0;
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    const int gridSize = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];

    if (inputDesc[0].type == DataType::kFLOAT)
    {
        constexpr int VPT = 16 / sizeof(float);
        constexpr int TPB = 256 / VPT;
        (layerNormKernel<float, TPB, VPT>)<<<gridSize, TPB, 0, stream>>>((const float *)inputs[0], (const float *)inputs[1], (const float *)inputs[2], (float *)outputs[0]);
    }
    else
    {
        constexpr int VPT = 16 / sizeof(half);
        constexpr int TPB = 256 / VPT;
        (layerNormKernel<half, TPB, VPT>)<<<gridSize, TPB, 0, stream>>>((const half *)inputs[0], (const half *)inputs[1], (const half *)inputs[2], (half *)outputs[0]);
    }
    return 0;
}

void LayerNormPlugin::destroy() noexcept
{
    WHERE_AM_I();
    delete this;
    return;
}

int32_t LayerNormPlugin::initialize() noexcept
{
    WHERE_AM_I();
    return 0;
}

void LayerNormPlugin::terminate() noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPlugin::getSerializationSize() const noexcept
{
    WHERE_AM_I();
    return sizeof(m_);
}

void LayerNormPlugin::serialize(void *buffer) const noexcept
{
    WHERE_AM_I();
    memcpy(buffer, &m_, sizeof(m_));
    return;
}

void LayerNormPlugin::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPlugin::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPlugin::getPluginType() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPlugin::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

void LayerNormPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
{
    WHERE_AM_I();
    return;
}

void LayerNormPlugin::detachFromContext() noexcept
{
    WHERE_AM_I();
    return;
}

// class LayerNormPluginCreator
PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

LayerNormPluginCreator::LayerNormPluginCreator()
{
    WHERE_AM_I();
    attr_.clear();
    attr_.emplace_back(PluginField("epsilon", nullptr, PluginFieldType::kFLOAT32, 1));
    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

LayerNormPluginCreator::~LayerNormPluginCreator()
{
    WHERE_AM_I();
}

IPluginV2 *LayerNormPluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    WHERE_AM_I();
    float                          epsilon = 1.0e-5f;
    std::map<std::string, float *> parameterMap {{"epsilon", &epsilon}};

    for (int i = 0; i < fc->nbFields; ++i)
    {
        if (parameterMap.find(fc->fields[i].name) != parameterMap.end())
        {
            *parameterMap[fc->fields[i].name] = *reinterpret_cast<const float *>(fc->fields[i].data);
        }
    }
    return new LayerNormPlugin(name, epsilon);
}

IPluginV2 *LayerNormPluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    WHERE_AM_I();
    return new LayerNormPlugin(name, serialData, serialLength);
}

void LayerNormPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPluginCreator::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPluginCreator::getPluginName() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPluginCreator::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

const PluginFieldCollection *LayerNormPluginCreator::getFieldNames() noexcept
{
    WHERE_AM_I();
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);

} // namespace nvinfer1
